#include "hip/hip_runtime.h"
#include "CudaRayTracing.cuh"
#include <exception>

#define PI (4 * atan(1))
#define PI2 (acos(0))
#define REFLECT_COEF 0.9

void RayIntensity(Ray* rays, int raysN, Cylinder* cyls, int cylsN, double* intens,
	double* TArr, double* nuArr, int TN, int nuN, double** kMatr);
__host__ __device__ double Inup(double nu, double T);
__host__ __device__ double Inu(double InuPrev, double nu, double dnu, double h, double k, double T);
__host__ __device__ double k(double T, int nu, double** kMatr, double* TArr, double* nuArr, int TN, int nuN);
__host__ __device__ double InuAbsorb(double InuPrev, double h, double k);


__device__ __host__ PairDouble GetIntersections(Ray ray, Cylinder cyl)
{
	double ac = cyl.R0, bc = cyl.R0;
	double y0 = ray.Start.Y, z0 = ray.Start.Z;
	double yd = ray.Direction.Y, zd = ray.Direction.Z;
	double b = (bc * bc * z0 * zd + ac * ac * y0 * yd);
	double a = (bc * bc * zd * zd + ac * ac * yd * yd);
	double c = (bc * bc * z0 * z0 + ac * ac * y0 * y0 - ac * ac * bc * bc);
	double d = b * b - a * c;
	double t1, t2;
	t1 = (-b + sqrt(d)) / a;
	t2 = (-b - sqrt(d)) / a;

	return { t1, t2 };
}

void RayIntensity(Ray* rays, int raysN, Cylinder* cyls, int cylsN, double* intens,
	double* TArr, double* nuArr, int TN, int nuN, double** kMatr)
{
	for (int nuId = 0; nuId < nuN - 1; nuId++)
	{
		for (int rId = 0; rId < raysN; rId++)
		{
			Ray ray = rays[rId];
			PairDouble inters = GetIntersections(ray, cyls[0]);
			double newStart = abs(inters.value1) > abs(inters.value2) ? inters.value1 : inters.value2;
			Ray newRay(ray.Point(newStart), ray.Direction);
			double dnu = nuArr[nuId + 1] - nuArr[nuId];
			double nu = (nuArr[nuId + 1] + nuArr[nuId]) / 2;
			double intensPrev = 0;
			int curCylI = 0;
			for (int i = 0; i < 2 * cylsN - 1; i++)
			{
				int nextCylI = (i + 1) < cylsN ? (i + 1) : cylsN - ((i + 1) % cylsN) - 1;

				inters = GetIntersections(newRay, cyls[nextCylI]);
				if (isnan(inters.value1) || isnan(inters.value2))
					continue;
				if (inters.value1 > 0 && inters.value2 > 0)
					newStart = inters.value1 < inters.value2 ? inters.value1 : inters.value2;
				else if (inters.value2 < 1e-12 && inters.value1 > 0)
					newStart = inters.value1;
				else if (inters.value2 > 0 && inters.value1 < 1e-12)
					newStart = inters.value2;
				else
					throw std::exception("Blb yf[eq");
				Cylinder curCyl = cyls[curCylI < nextCylI ? curCylI : nextCylI];
				double fk = k(curCyl.T, nuId, kMatr, TArr, nuArr, TN, nuN);
				double len = newRay.Start.Length(newRay.Point(newStart));
				double intens = Inu(intensPrev, nu, dnu, len, fk, curCyl.T);
				intensPrev = intens;
				curCylI = nextCylI;
				newRay = { newRay.Point(newStart), newRay.Direction };
			}
			intens[raysN * nuId + rId] = intensPrev;
		}
	}
}
		
void RayIntensityAbsorb(Ray * rays, int raysN, Cylinder * cyls, int cylsN, double* intens,
	double* TArr, double* nuArr, int TN, int nuN, double** kMatr, double* cylsEnergy)
{
	for (int nuId = 0; nuId < nuN - 1; nuId++)
	{
		for (int rId = 0; rId < raysN; rId++)
		{
			double intensPrev = intens[raysN * nuId + rId];
			Ray newRay = rays[rId];

			while (intensPrev > intens[raysN * nuId + rId] * 0.01)
			{
				PairDouble inters;
				int curCylI = 0;
				double newStart = 0;
				newRay.Direction = (-newRay.Direction).Reflect((-newRay.Start).Normalize());
				cylsEnergy[(cylsN * (nuN - 1) + nuId) * raysN + rId] += intensPrev * (1 - REFLECT_COEF);
				intensPrev *= REFLECT_COEF;
				for (int i = 0; i < 2 * cylsN - 1 && intensPrev > intens[raysN * nuId + rId] * 0.01; i++)
				{
					int nextCylI = (i + 1) < cylsN ? (i + 1) : cylsN - ((i + 1) % cylsN) - 1;

					inters = GetIntersections(newRay, cyls[nextCylI]);
					if (isnan(inters.value1) || isnan(inters.value2))
						continue;
					if (inters.value1 > 1e-12 && inters.value2 > 1e-12)
						newStart = inters.value1 < inters.value2 ? inters.value1 : inters.value2;
					else if (inters.value2 <= 1e-12 && inters.value1 > 1e-12)
						newStart = inters.value1;
					else if (inters.value2 > 1e-12 && inters.value1 <= 1e-12)
						newStart = inters.value2;
					else
						throw new std::exception("das");
					int ccI = curCylI < nextCylI ? curCylI : nextCylI;
					Cylinder curCyl = cyls[ccI];
					double fk = k(curCyl.T, nuId, kMatr, TArr, nuArr, TN, nuN);
					double len = newRay.Start.Length(newRay.Point(newStart));
					double inten = InuAbsorb(intensPrev, len, fk);
					cylsEnergy[(ccI * (nuN - 1) + nuId) * raysN + rId] += (intensPrev - inten);
					intensPrev = inten;
					curCylI = nextCylI;
					newRay = { newRay.Point(newStart), newRay.Direction };
					if (intensPrev < intens[raysN * nuId + rId] * 0.01)
					{
						if (i + 1 < 2 * cylsN - 1)
						{
							cylsEnergy[(nextCylI * (nuN - 1) + nuId) * raysN + rId] += intensPrev;
						}
						else
						{
							cylsEnergy[(curCylI * (nuN - 1) + nuId) * raysN + rId] += intensPrev;
						}
					}
				}
			}
		}
	}
}

__global__ void RayIntensityKernel(Ray* rays, Cylinder* cyls, int cylsN, double* intens,
	double* TArr, double* nuArr, int TN, int nuN, double** kMatr)
{
	int rId = threadIdx.x;
	int nuId = blockIdx.x;
	int raysN = blockDim.x;
	Ray ray = rays[rId];
	PairDouble inters = GetIntersections(ray, cyls[0]);
	double newStart = abs(inters.value1) > abs(inters.value2) ? inters.value1 : inters.value2;
	Ray newRay(ray.Point(newStart), ray.Direction);
	double dnu = nuArr[nuId + 1] - nuArr[nuId];
	double nu = (nuArr[nuId + 1] + nuArr[nuId]) / 2;
	double intensPrev = 0;
	int curCylI = 0;
	for (int i = 0; i < 2 * cylsN - 1; i++)
	{
		int nextCylI = (i + 1) < cylsN ? (i + 1) : cylsN - ((i + 1) % cylsN) - 1;

		inters = GetIntersections(newRay, cyls[nextCylI]);
		if (isnan(inters.value1) || isnan(inters.value2))
			continue;
		if (inters.value1 > 0 && inters.value2 > 0)
			newStart = inters.value1 < inters.value2 ? inters.value1 : inters.value2;
		else if (inters.value2 < 1e-18 && inters.value1 > 0)
			newStart = inters.value1;
		else if (inters.value2 > 0 && inters.value1 < 1e-18)
			newStart = inters.value2;
		else
			return;
		Cylinder curCyl = cyls[curCylI < nextCylI ? curCylI : nextCylI];
		double fk = k(curCyl.T, nuId, kMatr, TArr, nuArr, TN, nuN);
		double len = newRay.Start.Length(newRay.Point(newStart));
		double intens = Inu(intensPrev, nu, dnu, len, fk, curCyl.T);
		intensPrev = intens;
		curCylI = nextCylI;
		newRay = { newRay.Point(newStart), newRay.Direction };
	}
	intens[raysN * nuId + rId] = intensPrev;
}

__host__ __device__ double Inup(double nu, double T)
{
	double coef = 4.8e4 * nu / T;
	double e = exp(coef) - 1;
	return 1.47e6 * nu * nu * nu / e;
}

__host__ __device__ double Inu(double InuPrev, double nu, double dnu, double h, double k, double T)
{
	double e = exp(-k * h);
	return InuPrev * e + Inup(nu, T) * dnu * (1 - e);
}

__host__ __device__ double InuAbsorb(double InuPrev, double h, double k)
{
	double e = exp(-k * h);
	return InuPrev * e;
}

__host__ __device__ int binarySearch(double* arr, int low, int high, double x) {
	while (low < high) 
	{
		int mid = low + (high - low) / 2;
		if (arr[mid] < x) 
			low = mid + 1;
		else 
			high = mid - 1;
	}
	return low;
}

__host__ __device__ double k(double T, int nuId, double** kMatr, double* TArr, double* nuArr, int TN, int nuN)
{
	int TI = binarySearch(TArr, 0, TN, T);
	double kPrev = log(kMatr[TI][nuId]), kNext = log(kMatr[TI + 1][nuId]);
	double TPrev = log(TArr[TI]), TNext = log(TArr[TI + 1]);
	double kavg = (kPrev + (kNext - kPrev) / (TNext - TPrev) * (log(T) - TPrev));
	return exp(kavg);
}

hipError_t GetInitialIntensity(Lamp lamp, double* res, int n, int m)
{
	hipError_t cudaStatus;
	long raysN = n * m + 1;
	Ray* dev_rays;
	Ray* host_rays = new Ray[raysN];
	double* dev_intens;
	double* host_intens = new double[(lamp.nuN - 1) * raysN];
	double* devCylEnergy;
	double* hostCylEnergy = new double[(lamp.cylsN + 1) * (lamp.nuN - 1) * raysN] {0};
	Cylinder* dev_cyls;
	double** dev_kMatr, * dev_TArr, * dev_nuArr;
	Vector3 direction, start;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

#pragma region DevLightRay

	cudaStatus = hipMalloc(&dev_rays, raysN * sizeof(Ray));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc failed dev_rays!");
		goto Error;
	}

	for (int j = 1; j <= m; j++)
	{
		for (int i = 0; i < n; i++)
		{
			start = Vector3(0, 0, lamp.R);
			direction = Vector3(PI2 / (m + 1) * j, 2 * PI / n * i);
			Ray ray(start, direction);
			host_rays[i * m + j - 1] = ray;
		}
	}

	start = Vector3(0, 0, lamp.R);
	direction = Vector3(0, 0, 1);
	Ray ray(start, direction);
	host_rays[raysN - 1] = ray;

	cudaStatus = hipMalloc(&dev_rays, raysN * sizeof(Ray));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMemcpy failed! ray start");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_rays, host_rays, sizeof(Ray) * raysN, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("hipMemcpy failed! ray start");
		goto Error;
	}
#pragma endregion

#pragma region DevIntense
	cudaStatus = hipMalloc(&dev_intens, sizeof(double) * (lamp.nuN - 1) * raysN);
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc failed dev_intens!");
		goto Error;
	}
#pragma endregion

	cudaStatus = hipMalloc(&devCylEnergy, sizeof(double) * (lamp.cylsN + 1) * (lamp.nuN - 1) * raysN);
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc failed dev_intens!");
		goto Error;
	}

#pragma region DevCylinders
	cudaStatus = hipMalloc(&dev_cyls, lamp.cylsN * sizeof(Cylinder));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc failed dev_cyls!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_cyls, lamp.Cylinders, lamp.cylsN * sizeof(Cylinder), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("hipMemcpy failed!");
		goto Error;
	}
#pragma endregion

#pragma region DevKMatr
	cudaStatus = hipMalloc(&dev_kMatr, lamp.TN * sizeof(double*));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc failed dev_kMatr!");
		goto Error;
	}

	for (int i = 0; i < lamp.TN; i++)
	{
		double* dev_kMatrRow;
		cudaStatus = hipMalloc(&dev_kMatrRow, (lamp.nuN - 1) * sizeof(double));
		if (cudaStatus != hipSuccess)
		{
			printf("hipMalloc failed dev_kMatr %d!", i);
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_kMatrRow, lamp.kMatr[i], sizeof(double) * (lamp.nuN - 1), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			printf("hipMemcpy failed! dev_kMatr %d", i);
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_kMatr + i, &dev_kMatrRow, sizeof(double*), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			printf("hipMemcpy failed! dev_kMatrRow %d", i);
			goto Error;
		}
	}
#pragma endregion

#pragma region DevTArr
	cudaStatus = hipMalloc(&dev_TArr, lamp.TN * sizeof(double));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc failed dev_TArr!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_TArr, lamp.TArr, sizeof(double) * lamp.TN, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("hipMemcpy failed! dev_TArr");
		goto Error;
	}
#pragma endregion

#pragma region DevNuArr
	cudaStatus = hipMalloc(&dev_nuArr, lamp.nuN * sizeof(double));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc failed dev_nuArr!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_nuArr, lamp.nuArr, sizeof(double) * lamp.nuN, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("hipMemcpy failed! dev_nuArr");
		goto Error;
	}
#pragma endregion

	RayIntensity(host_rays, raysN, lamp.Cylinders, lamp.cylsN, host_intens,
		lamp.TArr, lamp.nuArr, lamp.TN, lamp.nuN, lamp.kMatr);

	double dtetha = PI2 / (m + 1);
	double dphi = 2 * PI / n;
	for (int l = 0; l < lamp.nuN - 1; l++)
	{
		double sum = 0;
		for (int r = 0; r < raysN; r++)
		{
			double x = host_rays[r].Direction.X;
			double y = host_rays[r].Direction.Y;
			double z = host_rays[r].Direction.Z;
			double tetha = atan(sqrt(x * x + y * y) / z);
			host_intens[l * raysN + r] *= dphi * dtetha * sin(tetha) * cos(tetha);
			//printf("%.3g ", host_intens[l * raysN + r]);
		}
		//printf("\n");
	}

	printf("\n");
	RayIntensityAbsorb(host_rays, raysN, lamp.Cylinders, lamp.cylsN, host_intens,
		lamp.TArr, lamp.nuArr, lamp.TN, lamp.nuN, lamp.kMatr, hostCylEnergy);
	double prevR = 0;
	for (int l = 0; l < lamp.nuN - 1; l++)
	{
		double nuAvg = (lamp.nuArr[l] + lamp.nuArr[l + 1]) / 2;
		
		for (int i = lamp.cylsN - 1; i >= 0; i--)
		{
			double K = k(lamp.Cylinders[i].T, l, lamp.kMatr, lamp.TArr, lamp.nuArr, lamp.TN, lamp.nuN);
			double c = 3e8;
			double q3 = 0;
			for (int r = 0; r < raysN; r++)
			{
				q3 += hostCylEnergy[(i * (lamp.nuN - 1) + l) * raysN + r];
			}
			double qp = q3 * 2 * lamp.R / (lamp.Cylinders[i].R0 * lamp.Cylinders[i].R0 - prevR * prevR);
			prevR = lamp.Cylinders[i].R0;
			double Fl = Inup(nuAvg, lamp.Cylinders[i].T) * 4 * PI * K - qp;
			printf("%.3g ", Fl);
		}
		printf("\n");
	}


	//for (int l = 0; l < lamp.nuN - 1; l++)
	//{
	//	for (int i = 0; i < lamp.cylsN + 1; i++)
	//	{
	//		double sum = 0;
	//		for (int r = 0; r < raysN; r++)
	//		{
	//			sum += hostCylEnergy[(i * (lamp.nuN - 1) + l) * raysN + r];
	//		}
	//		printf("%.3g ", sum);
	//	}
	//	printf("\n");
	//}
	/*RayIntensityKernel<<<lamp.nuN - 1, raysN>>>(dev_rays, dev_cyls, lamp.cylsN, dev_intens,
		dev_TArr, dev_nuArr, lamp.TN, lamp.nuN, dev_kMatr);*/

	//Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching RayIntensityKernel!\n%s\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}
	//for (int j = 0; j < lamp.nuN; j++)
	//{
	//	printf("%.4e\n", lamp.nuArr[j]);
	//	for (int i = 0; i < n * m + 1; i++)
	//		printf("%g ", host_intens[j * (n * m + 1) + i]);
	//	printf("\n");
	//}
	// Copy output vector from GPU buffer to host memory.
	//cudaStatus = hipMemcpy(host_intens, dev_intens, sizeof(double) * (lamp.nuN - 1) * raysN, hipMemcpyDeviceToHost);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!");
	//	goto Error;
	//}

Error:
	return cudaStatus;
}

